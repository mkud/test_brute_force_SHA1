
#include "hip/hip_runtime.h"


#include <stdio.h>

#define __u32 unsigned int 
#define __u8 unsigned char

hipError_t addWithCuda(__u32 SHA1Sum[], __u8 SHA1Data[], int *retVal);


#define	FETCH_32(p)							\
	(((__u32)*((const __u8 *)(p) + 3)) |			\
	(((__u32)*((const __u8 *)(p) + 2)) << 8) |		\
	(((__u32)*((const __u8 *)(p) + 1)) << 16) |		\
	(((__u32)*((const __u8 *)(p))) << 24))

/* Constants from FIPS 180-1 */
#define	K_00_19		0x5a827999UL
#define	K_20_39		0x6ed9eba1UL
#define	K_40_59		0x8f1bbcdcUL
#define	K_60_79		0xca62c1d6UL

/* F, G, H and I are basic SHA1 functions. */
#define	F(b, c, d)	((((c) ^ (d)) & (b)) ^ (d))
#define	G(b, c, d)	((b) ^ (c) ^ (d))
#define	H(b, c, d)	(((b) & (c)) | (((b) | (c)) & (d)))

/* ROTATE_LEFT rotates x left n bits. */
#define	ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32 - (n))))

/* R, R1-R4 are macros used during each transformation round. */
#define R(f, k, v, w, x, y, z, i) {				\
	(v) = ROTATE_LEFT(w, 5) + f(x, y, z) + (v) + (i) + (k);	\
	(x) = ROTATE_LEFT(x, 30);				\
}

#define	R1(v, w, x, y, z, i)	R(F, K_00_19, v, w, x, y, z, i)
#define	R2(v, w, x, y, z, i)	R(G, K_20_39, v, w, x, y, z, i)
#define	R3(v, w, x, y, z, i)	R(H, K_40_59, v, w, x, y, z, i)
#define	R4(v, w, x, y, z, i)	R(G, K_60_79, v, w, x, y, z, i)

#define	WUPDATE(p, q, r, s) {		\
	(p) = ((q) ^ (r) ^ (s) ^ (p));	\
	(p) = ROTATE_LEFT(p, 1);	\
}

void SHA1Init(__u32 mass[]) {
	mass[0] = 0x67452301UL;
	mass[1] = 0xefcdab89UL;
	mass[2] = 0x98badcfeUL;
	mass[3] = 0x10325476UL;
	mass[4] = 0xc3d2e1f0UL;
}

void
sha_transformINIT(__u32 mass[], __u8 block[])
{
	__u32 a = mass[0];
	__u32 b = mass[1];
	__u32 c = mass[2];
	__u32 d = mass[3];
	__u32 e = mass[4];

	/* Register (instead of array) is a win in most cases */
	__u32 w0, w1, w2, w3, w4, w5, w6, w7;
	__u32 w8, w9, w10, w11, w12, w13, w14, w15;

	w15 = FETCH_32(block + 60);
	w14 = FETCH_32(block + 56);
	w13 = FETCH_32(block + 52);
	w12 = FETCH_32(block + 48);
	w11 = FETCH_32(block + 44);
	w10 = FETCH_32(block + 40);
	w9 = FETCH_32(block + 36);
	w8 = FETCH_32(block + 32);
	w7 = FETCH_32(block + 28);
	w6 = FETCH_32(block + 24);
	w5 = FETCH_32(block + 20);
	w4 = FETCH_32(block + 16);
	w3 = FETCH_32(block + 12);
	w2 = FETCH_32(block + 8);
	w1 = FETCH_32(block + 4);
	w0 = FETCH_32(block + 0);

	/* Round 1 */
	R1(e, a, b, c, d, w0);		/*  0 */
	R1(d, e, a, b, c, w1);		/*  1 */
	R1(c, d, e, a, b, w2);		/*  2 */
	R1(b, c, d, e, a, w3);		/*  3 */
	R1(a, b, c, d, e, w4);		/*  4 */
	R1(e, a, b, c, d, w5);		/*  5 */
	R1(d, e, a, b, c, w6);		/*  6 */
	R1(c, d, e, a, b, w7);		/*  7 */
	R1(b, c, d, e, a, w8);		/*  8 */
	R1(a, b, c, d, e, w9);		/*  9 */
	R1(e, a, b, c, d, w10);		/* 10 */
	R1(d, e, a, b, c, w11);		/* 11 */
	R1(c, d, e, a, b, w12);		/* 12 */
	R1(b, c, d, e, a, w13);		/* 13 */
	R1(a, b, c, d, e, w14);		/* 14 */
	R1(e, a, b, c, d, w15);		/* 15 */
	WUPDATE(w0, w13, w8, w2);	R1(d, e, a, b, c, w0);		/* 16 */
	WUPDATE(w1, w14, w9, w3);	R1(c, d, e, a, b, w1);		/* 17 */
	WUPDATE(w2, w15, w10, w4);	R1(b, c, d, e, a, w2);		/* 18 */
	WUPDATE(w3, w0, w11, w5);	R1(a, b, c, d, e, w3);		/* 19 */

	/* Round 2 */
	WUPDATE(w4, w1, w12, w6);	R2(e, a, b, c, d, w4);		/* 20 */
	WUPDATE(w5, w2, w13, w7);	R2(d, e, a, b, c, w5);		/* 21 */
	WUPDATE(w6, w3, w14, w8);	R2(c, d, e, a, b, w6);		/* 22 */
	WUPDATE(w7, w4, w15, w9);	R2(b, c, d, e, a, w7);		/* 23 */
	WUPDATE(w8, w5, w0, w10);	R2(a, b, c, d, e, w8);		/* 24 */
	WUPDATE(w9, w6, w1, w11);	R2(e, a, b, c, d, w9);		/* 25 */
	WUPDATE(w10, w7, w2, w12);	R2(d, e, a, b, c, w10);		/* 26 */
	WUPDATE(w11, w8, w3, w13);	R2(c, d, e, a, b, w11);		/* 27 */
	WUPDATE(w12, w9, w4, w14);	R2(b, c, d, e, a, w12);		/* 28 */
	WUPDATE(w13, w10, w5, w15);	R2(a, b, c, d, e, w13);		/* 29 */
	WUPDATE(w14, w11, w6, w0);	R2(e, a, b, c, d, w14);		/* 30 */
	WUPDATE(w15, w12, w7, w1);	R2(d, e, a, b, c, w15);		/* 31 */
	WUPDATE(w0, w13, w8, w2);	R2(c, d, e, a, b, w0);		/* 32 */
	WUPDATE(w1, w14, w9, w3);	R2(b, c, d, e, a, w1);		/* 33 */
	WUPDATE(w2, w15, w10, w4);	R2(a, b, c, d, e, w2);		/* 34 */
	WUPDATE(w3, w0, w11, w5);	R2(e, a, b, c, d, w3);		/* 35 */
	WUPDATE(w4, w1, w12, w6);	R2(d, e, a, b, c, w4);		/* 36 */
	WUPDATE(w5, w2, w13, w7);	R2(c, d, e, a, b, w5);		/* 37 */
	WUPDATE(w6, w3, w14, w8);	R2(b, c, d, e, a, w6);		/* 38 */
	WUPDATE(w7, w4, w15, w9);	R2(a, b, c, d, e, w7);		/* 39 */

	/* Round 3 */
	WUPDATE(w8, w5, w0, w10);	R3(e, a, b, c, d, w8);		/* 40 */
	WUPDATE(w9, w6, w1, w11);	R3(d, e, a, b, c, w9);		/* 41 */
	WUPDATE(w10, w7, w2, w12);	R3(c, d, e, a, b, w10);		/* 42 */
	WUPDATE(w11, w8, w3, w13);	R3(b, c, d, e, a, w11);		/* 43 */
	WUPDATE(w12, w9, w4, w14);	R3(a, b, c, d, e, w12);		/* 44 */
	WUPDATE(w13, w10, w5, w15);	R3(e, a, b, c, d, w13);		/* 45 */
	WUPDATE(w14, w11, w6, w0);	R3(d, e, a, b, c, w14);		/* 46 */
	WUPDATE(w15, w12, w7, w1);	R3(c, d, e, a, b, w15);		/* 47 */
	WUPDATE(w0, w13, w8, w2);	R3(b, c, d, e, a, w0);		/* 48 */
	WUPDATE(w1, w14, w9, w3);	R3(a, b, c, d, e, w1);		/* 49 */
	WUPDATE(w2, w15, w10, w4);	R3(e, a, b, c, d, w2);		/* 50 */
	WUPDATE(w3, w0, w11, w5);	R3(d, e, a, b, c, w3);		/* 51 */
	WUPDATE(w4, w1, w12, w6);	R3(c, d, e, a, b, w4);		/* 52 */
	WUPDATE(w5, w2, w13, w7);	R3(b, c, d, e, a, w5);		/* 53 */
	WUPDATE(w6, w3, w14, w8);	R3(a, b, c, d, e, w6);		/* 54 */
	WUPDATE(w7, w4, w15, w9);	R3(e, a, b, c, d, w7);		/* 55 */
	WUPDATE(w8, w5, w0, w10);	R3(d, e, a, b, c, w8);		/* 56 */
	WUPDATE(w9, w6, w1, w11);	R3(c, d, e, a, b, w9);		/* 57 */
	WUPDATE(w10, w7, w2, w12);	R3(b, c, d, e, a, w10);		/* 58 */
	WUPDATE(w11, w8, w3, w13);	R3(a, b, c, d, e, w11);		/* 59 */

	WUPDATE(w12, w9, w4, w14);	R4(e, a, b, c, d, w12);		/* 60 */
	WUPDATE(w13, w10, w5, w15);	R4(d, e, a, b, c, w13);		/* 61 */
	WUPDATE(w14, w11, w6, w0);	R4(c, d, e, a, b, w14);		/* 62 */
	WUPDATE(w15, w12, w7, w1);	R4(b, c, d, e, a, w15);		/* 63 */
	WUPDATE(w0, w13, w8, w2);	R4(a, b, c, d, e, w0);		/* 64 */
	WUPDATE(w1, w14, w9, w3);	R4(e, a, b, c, d, w1);		/* 65 */
	WUPDATE(w2, w15, w10, w4);	R4(d, e, a, b, c, w2);		/* 66 */
	WUPDATE(w3, w0, w11, w5);	R4(c, d, e, a, b, w3);		/* 67 */
	WUPDATE(w4, w1, w12, w6);	R4(b, c, d, e, a, w4);		/* 68 */
	WUPDATE(w5, w2, w13, w7);	R4(a, b, c, d, e, w5);		/* 69 */
	WUPDATE(w6, w3, w14, w8);	R4(e, a, b, c, d, w6);		/* 70 */
	WUPDATE(w7, w4, w15, w9);	R4(d, e, a, b, c, w7);		/* 71 */
	WUPDATE(w8, w5, w0, w10);	R4(c, d, e, a, b, w8);		/* 72 */
	WUPDATE(w9, w6, w1, w11);	R4(b, c, d, e, a, w9);		/* 73 */
	WUPDATE(w10, w7, w2, w12);	R4(a, b, c, d, e, w10);		/* 74 */
	WUPDATE(w11, w8, w3, w13);	R4(e, a, b, c, d, w11);		/* 75 */
	WUPDATE(w12, w9, w4, w14);	R4(d, e, a, b, c, w12);		/* 76 */
	WUPDATE(w13, w10, w5, w15);	R4(c, d, e, a, b, w13);		/* 77 */
	WUPDATE(w14, w11, w6, w0);	R4(b, c, d, e, a, w14);		/* 78 */
	WUPDATE(w15, w12, w7, w1);	R4(a, b, c, d, e, w15);		/* 79 */

	mass[0] += a;
	mass[1] += b;
	mass[2] += c;
	mass[3] += d;
	mass[4] += e;
}

__constant__ __u32 g_cSHA1baseCode[5];

__global__ void
sha_transform(int *ret_block, __u32 *in_block)
{
	__u32 *block = in_block + ((blockIdx.x << 5) + threadIdx.x) * 16;
	__u8 *pos_cur;

	__u32 a, b, c, d, e;
	/* Register (instead of array) is a win in most cases */
	__u32 w0, w1, w2, w3, w4, w5, w6, w7;
	__u32 w8, w9, w10, w11, w12, w13, w14, w15;
	__u32 wi[16];
	wi[0] = block[0];
	wi[1] = block[1];
	wi[2] = block[2];
	wi[3] = block[3];
	wi[4] = block[4];
	wi[5] = block[5];
	wi[6] = block[6];
	wi[7] = block[7];
	wi[8] = block[8];
	wi[9] = block[9];
	wi[10] = block[10];
	wi[11] = block[11];
	wi[12] = block[12];
	wi[13] = block[13];
	wi[14] = block[14];
	wi[15] = block[15];



	while (true) {
		pos_cur = (__u8*)wi;
		while (true) {
			if ((*pos_cur) == 126) {
				(*pos_cur) = 33;
				pos_cur++;
				continue;
			}
			(*pos_cur)++;

			a = g_cSHA1baseCode[0];
			b = g_cSHA1baseCode[1];
			c = g_cSHA1baseCode[2];
			d = g_cSHA1baseCode[3];
			e = g_cSHA1baseCode[4];

/*			w15 = FETCH_32((__u8 *)(block + 15));
			w14 = FETCH_32((__u8 *)(block + 14));
			w13 = FETCH_32((__u8 *)(block + 13));
			w12 = FETCH_32((__u8 *)(block + 12));
			w11 = FETCH_32((__u8 *)(block + 11));
			w10 = FETCH_32((__u8 *)(block + 10));
			w9 = FETCH_32((__u8 *)(block + 9));
			w8 = FETCH_32((__u8 *)(block + 8));
			w7 = FETCH_32((__u8 *)(block + 7));
			w6 = FETCH_32((__u8 *)(block + 6));
			w5 = FETCH_32((__u8 *)(block + 5));
			w4 = FETCH_32((__u8 *)(block + 4));
			w3 = FETCH_32((__u8 *)(block + 3));
			w2 = FETCH_32((__u8 *)(block + 2));
			w1 = FETCH_32((__u8 *)(block + 1));
			w0 = FETCH_32((__u8 *)(block + 0));
			*/
			w15 = wi[15];
			w14 = wi[14];
			w13 = wi[13];
			w12 = wi[12];
			w11 = wi[11];
			w10 = wi[10];
			w9 = wi[9];
			w8 = wi[8];
			w7 = wi[7];
			w6 = wi[6];
			w5 = wi[5];
			w4 = wi[4];
			w3 = wi[3];
			w2 = wi[2];
			w1 = wi[1];
			w0 = wi[0];

			/* Round 1 */
			R1(e, a, b, c, d, w0);		/*  0 */
			R1(d, e, a, b, c, w1);		/*  1 */
			R1(c, d, e, a, b, w2);		/*  2 */
			R1(b, c, d, e, a, w3);		/*  3 */
			R1(a, b, c, d, e, w4);		/*  4 */
			R1(e, a, b, c, d, w5);		/*  5 */
			R1(d, e, a, b, c, w6);		/*  6 */
			R1(c, d, e, a, b, w7);		/*  7 */
			R1(b, c, d, e, a, w8);		/*  8 */
			R1(a, b, c, d, e, w9);		/*  9 */
			R1(e, a, b, c, d, w10);		/* 10 */
			R1(d, e, a, b, c, w11);		/* 11 */
			R1(c, d, e, a, b, w12);		/* 12 */
			R1(b, c, d, e, a, w13);		/* 13 */
			R1(a, b, c, d, e, w14);		/* 14 */
			R1(e, a, b, c, d, w15);		/* 15 */
			WUPDATE(w0, w13, w8, w2);	R1(d, e, a, b, c, w0);		/* 16 */
			WUPDATE(w1, w14, w9, w3);	R1(c, d, e, a, b, w1);		/* 17 */
			WUPDATE(w2, w15, w10, w4);	R1(b, c, d, e, a, w2);		/* 18 */
			WUPDATE(w3, w0, w11, w5);	R1(a, b, c, d, e, w3);		/* 19 */

			/* Round 2 */
			WUPDATE(w4, w1, w12, w6);	R2(e, a, b, c, d, w4);		/* 20 */
			WUPDATE(w5, w2, w13, w7);	R2(d, e, a, b, c, w5);		/* 21 */
			WUPDATE(w6, w3, w14, w8);	R2(c, d, e, a, b, w6);		/* 22 */
			WUPDATE(w7, w4, w15, w9);	R2(b, c, d, e, a, w7);		/* 23 */
			WUPDATE(w8, w5, w0, w10);	R2(a, b, c, d, e, w8);		/* 24 */
			WUPDATE(w9, w6, w1, w11);	R2(e, a, b, c, d, w9);		/* 25 */
			WUPDATE(w10, w7, w2, w12);	R2(d, e, a, b, c, w10);		/* 26 */
			WUPDATE(w11, w8, w3, w13);	R2(c, d, e, a, b, w11);		/* 27 */
			WUPDATE(w12, w9, w4, w14);	R2(b, c, d, e, a, w12);		/* 28 */
			WUPDATE(w13, w10, w5, w15);	R2(a, b, c, d, e, w13);		/* 29 */
			WUPDATE(w14, w11, w6, w0);	R2(e, a, b, c, d, w14);		/* 30 */
			WUPDATE(w15, w12, w7, w1);	R2(d, e, a, b, c, w15);		/* 31 */
			WUPDATE(w0, w13, w8, w2);	R2(c, d, e, a, b, w0);		/* 32 */
			WUPDATE(w1, w14, w9, w3);	R2(b, c, d, e, a, w1);		/* 33 */
			WUPDATE(w2, w15, w10, w4);	R2(a, b, c, d, e, w2);		/* 34 */
			WUPDATE(w3, w0, w11, w5);	R2(e, a, b, c, d, w3);		/* 35 */
			WUPDATE(w4, w1, w12, w6);	R2(d, e, a, b, c, w4);		/* 36 */
			WUPDATE(w5, w2, w13, w7);	R2(c, d, e, a, b, w5);		/* 37 */
			WUPDATE(w6, w3, w14, w8);	R2(b, c, d, e, a, w6);		/* 38 */
			WUPDATE(w7, w4, w15, w9);	R2(a, b, c, d, e, w7);		/* 39 */

			/* Round 3 */
			WUPDATE(w8, w5, w0, w10);	R3(e, a, b, c, d, w8);		/* 40 */
			WUPDATE(w9, w6, w1, w11);	R3(d, e, a, b, c, w9);		/* 41 */
			WUPDATE(w10, w7, w2, w12);	R3(c, d, e, a, b, w10);		/* 42 */
			WUPDATE(w11, w8, w3, w13);	R3(b, c, d, e, a, w11);		/* 43 */
			WUPDATE(w12, w9, w4, w14);	R3(a, b, c, d, e, w12);		/* 44 */
			WUPDATE(w13, w10, w5, w15);	R3(e, a, b, c, d, w13);		/* 45 */
			WUPDATE(w14, w11, w6, w0);	R3(d, e, a, b, c, w14);		/* 46 */
			WUPDATE(w15, w12, w7, w1);	R3(c, d, e, a, b, w15);		/* 47 */
			WUPDATE(w0, w13, w8, w2);	R3(b, c, d, e, a, w0);		/* 48 */
			WUPDATE(w1, w14, w9, w3);	R3(a, b, c, d, e, w1);		/* 49 */
			WUPDATE(w2, w15, w10, w4);	R3(e, a, b, c, d, w2);		/* 50 */
			WUPDATE(w3, w0, w11, w5);	R3(d, e, a, b, c, w3);		/* 51 */
			WUPDATE(w4, w1, w12, w6);	R3(c, d, e, a, b, w4);		/* 52 */
			WUPDATE(w5, w2, w13, w7);	R3(b, c, d, e, a, w5);		/* 53 */
			WUPDATE(w6, w3, w14, w8);	R3(a, b, c, d, e, w6);		/* 54 */
			WUPDATE(w7, w4, w15, w9);	R3(e, a, b, c, d, w7);		/* 55 */
			WUPDATE(w8, w5, w0, w10);	R3(d, e, a, b, c, w8);		/* 56 */
			WUPDATE(w9, w6, w1, w11);	R3(c, d, e, a, b, w9);		/* 57 */
			WUPDATE(w10, w7, w2, w12);	R3(b, c, d, e, a, w10);		/* 58 */
			WUPDATE(w11, w8, w3, w13);	R3(a, b, c, d, e, w11);		/* 59 */

			WUPDATE(w12, w9, w4, w14);	R4(e, a, b, c, d, w12);		/* 60 */
			WUPDATE(w13, w10, w5, w15);	R4(d, e, a, b, c, w13);		/* 61 */
			WUPDATE(w14, w11, w6, w0);	R4(c, d, e, a, b, w14);		/* 62 */
			WUPDATE(w15, w12, w7, w1);	R4(b, c, d, e, a, w15);		/* 63 */
			WUPDATE(w0, w13, w8, w2);	R4(a, b, c, d, e, w0);		/* 64 */
			WUPDATE(w1, w14, w9, w3);	R4(e, a, b, c, d, w1);		/* 65 */
			WUPDATE(w2, w15, w10, w4);	R4(d, e, a, b, c, w2);		/* 66 */
			WUPDATE(w3, w0, w11, w5);	R4(c, d, e, a, b, w3);		/* 67 */
			WUPDATE(w4, w1, w12, w6);	R4(b, c, d, e, a, w4);		/* 68 */
			WUPDATE(w5, w2, w13, w7);	R4(a, b, c, d, e, w5);		/* 69 */
			WUPDATE(w6, w3, w14, w8);	R4(e, a, b, c, d, w6);		/* 70 */
			WUPDATE(w7, w4, w15, w9);	R4(d, e, a, b, c, w7);		/* 71 */
			WUPDATE(w8, w5, w0, w10);	R4(c, d, e, a, b, w8);		/* 72 */
			WUPDATE(w9, w6, w1, w11);	R4(b, c, d, e, a, w9);		/* 73 */
			WUPDATE(w10, w7, w2, w12);	R4(a, b, c, d, e, w10);		/* 74 */
			WUPDATE(w11, w8, w3, w13);	R4(e, a, b, c, d, w11);		/* 75 */
			WUPDATE(w12, w9, w4, w14);	R4(d, e, a, b, c, w12);		/* 76 */
			WUPDATE(w13, w10, w5, w15);	R4(c, d, e, a, b, w13);		/* 77 */
			WUPDATE(w14, w11, w6, w0);	R4(b, c, d, e, a, w14);		/* 78 */
			WUPDATE(w15, w12, w7, w1);	R4(a, b, c, d, e, w15);		/* 79 */
			a += g_cSHA1baseCode[0];
			
			if (((a) == 0)) {
				b += g_cSHA1baseCode[1];
				if ((b & 0xffff0000) == 0) {
					block[0] = wi[0];
					block[1] = wi[1];
					block[2] = wi[2];
					block[3] = wi[3];
					block[4] = wi[4];
					block[5] = wi[5];
					block[6] = wi[6];
					block[7] = wi[7];
					block[8] = wi[8];
					block[9] = wi[9];
					block[10] = wi[10];
					block[11] = wi[11];
					block[12] = wi[12];
					block[13] = wi[13];
					block[14] = wi[14];
					block[15] = wi[15];

					(*ret_block) = (blockIdx.x << 5) + threadIdx.x;
					return;
				}
			}
			if ((*ret_block) != -1)
				return;

			break;
		}
	}
}
#include <random>
#include <chrono>

int main(int argc, char **argv)
{
	__u32 buf[5];
	SHA1Init(buf);
	sha_transformINIT(buf, (unsigned char*)argv[1]);
	
	std::default_random_engine generator(std::chrono::system_clock::now().time_since_epoch().count());
	std::uniform_int_distribution<int> distribution(33, 126);

	unsigned char data[2048 * 64];
	for (int j = 0; j < 2048; j++) {
		for (int i = 0; i < 56; i++)
			data[j*64 + i] = distribution(generator);

		data[j * 64 + 52] = 0x80;
		memset(j * 64 + data + 56, 0, 8);
		data[j * 64 + 61] = 0x03;
		data[j * 64 + 60] = 0xb8;
	}

	int iRetVal = -1;
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(buf, data, &iRetVal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	for (int i = 0; i < 13; i++)
		for (int j = 3; j >= 0; j--)
			printf("%x ", data[iRetVal * 64 + (i << 2) + j]);
	printf("%x %x %x", data[iRetVal * 64 + 55], data[iRetVal * 64 + 54], data[iRetVal * 64 + 53]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(__u32 SHA1Sum[], __u8 SHA1Data[], int *retVal)
{
	__u32 *dev_SHA1Data = 0;
	int *dev_retVal = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_SHA1Data, 2048 * 64 * sizeof(__u8));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_retVal, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(g_cSHA1baseCode), SHA1Sum, 5 * sizeof(__u32));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyToSymbol failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_SHA1Data, SHA1Data, 2048 * 64 * sizeof(__u8), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_retVal, retVal, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch a kernel on the GPU with one thread for each element.
	sha_transform <<<64, 32>>>(dev_retVal, dev_SHA1Data);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(SHA1Data, dev_SHA1Data, 2048 * 64 * sizeof(__u8), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(retVal, dev_retVal, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
    hipFree(dev_SHA1Data);
	hipFree(dev_retVal);

    return cudaStatus;
}
